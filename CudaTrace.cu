#include <iostream>
#include <ray_funcs.h>	

__host__ void cuda_trace(int num_threads, tira::camera cam, tira::triangle* tri, spheres* s1, spheres bound_sph, lights* l1, plane p, int res_x, int res_y, int tri_count, int num_sph, int num_lights, int num_planes, int num_mesh, float* image_mat)
{
	tira::camera *device_cam;
	tira::triangle* device_tri;
	spheres *device_sph, *dev_bound_sph; 
	lights* device_lights;
	plane* dev_p;
	float* dev_image_mat;
	hipError_t err;

	//malloc

	err = hipMalloc((void**)&device_cam, sizeof(tira::camera));
	HANDLE_ERROR(err);

	err = hipMalloc((void**)&device_tri, tri_count * sizeof(tira::triangle));
	HANDLE_ERROR(err);

	err = hipMalloc(&device_sph, num_sph * sizeof(spheres));
	HANDLE_ERROR(err);

	err = hipMalloc(&dev_bound_sph, sizeof(spheres));
	HANDLE_ERROR(err);

	err = hipMalloc(&device_lights, num_lights * sizeof(lights));
	HANDLE_ERROR(err);

	err = hipMalloc(&dev_p, sizeof(plane));
	HANDLE_ERROR(err);

	err = hipMalloc(&dev_image_mat, res_x * res_y * 3 * sizeof(float));
	HANDLE_ERROR(err);

	//memcpy

	err = hipMemcpy(dev_image_mat, image_mat, res_x * res_y * 3 * sizeof(float), hipMemcpyHostToDevice);
	HANDLE_ERROR(err);

	err = hipMemcpy(device_cam, &cam, sizeof(tira::camera), hipMemcpyHostToDevice);
	HANDLE_ERROR(err);

	err = hipMemcpy(device_sph, s1, num_sph * sizeof(spheres), hipMemcpyHostToDevice);
	HANDLE_ERROR(err);

	err = hipMemcpy(dev_bound_sph, &bound_sph, sizeof(spheres), hipMemcpyHostToDevice);
	HANDLE_ERROR(err);

	err = hipMemcpy(device_lights, l1, num_lights * sizeof(lights), hipMemcpyHostToDevice);
	HANDLE_ERROR(err);

	err = hipMemcpy(dev_p, &p, sizeof(plane), hipMemcpyHostToDevice);
	HANDLE_ERROR(err);

	err = hipMemcpy(device_tri, tri, tri_count * sizeof(tira::triangle), hipMemcpyHostToDevice);
	HANDLE_ERROR(err);

	dim3 blockSize(16, 16); // Each block has 16x16 threads
	dim3 gridSize(63,63); // Number of blocks needed

	raytrace_loop_cuda << < gridSize,blockSize >>>(device_cam, device_tri, device_sph, dev_bound_sph, device_lights, dev_p, res_x, res_y, tri_count, num_sph, num_lights, num_planes, num_mesh, dev_image_mat);
	err = hipDeviceSynchronize();
	HANDLE_ERROR(err);

	err = hipMemcpy(image_mat, dev_image_mat, res_x * res_y * 3 * sizeof(float), hipMemcpyDeviceToHost);
	HANDLE_ERROR(err);

	hipFree(dev_image_mat);
	hipFree(device_cam);
	hipFree(device_tri);
	hipFree(device_sph);
	hipFree(dev_bound_sph);
	hipFree(device_lights);
	hipFree(dev_p);
	//hipMemcpy(y, x, sizeof(int), hipMemcpyDeviceToHost);
}